#include "hip/hip_runtime.h"
//#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <float.h>
#include <string.h>
#include <math.h>

typedef struct coord_s {
	float x;
        float y;
} coord;

typedef struct node_s {
	coord coord;
	int label;
} node;

typedef struct cluster_s {
	int num_nodes;
	node* node;
} cluster;

void read_nodes(int num_nodes, node* nodes, FILE* f) {
	for (int i = 0; i < num_nodes; i++) {
		node* n = &(nodes[i]);
		if (fscanf(f, "%f %f\n", &(n->coord.x), &(n->coord.y))) {
			n->label = i;
			continue;
		} else {
			printf("read line %d failed.\n", i);
			return;
		}

	}
	for (int i = 0; i < num_nodes; i++) {
		printf("%d: x = %f, y = %f\n", nodes[i].label, nodes[i].coord.x, nodes[i].coord.y);
	}

}

int read_data(node** nodes, const char* file) {
	
	FILE* f = fopen (file, "r");

	printf("file opend.\n");	
	int num_node;
	if (fscanf(f, "%5d\n", &num_node) == 0) {
		printf("read inputfile failed.\n");
		return 0;
	}
	printf("num_node = %d\n", num_node);
	
	*nodes = (node*)malloc(num_node*sizeof(node));
	if (*nodes) {
		read_nodes(num_node, *nodes, f);
	//	free(nodes);
	} else {
		printf("malloc failed.\n");
		return 0;
	}
	
	fclose(f);
	printf("file closed.\n");
	
	return num_node;
}		

//__device__ 
float dist(const node* n1, const node* n2) {
	return sqrt(pow(n1->coord.x - n2->coord.x, 2) + pow(n1->coord.y - n2->coord.y, 2));
}

//__device__ 
void generate_dist_matrix(int num_nodes, node* nodes, float* dist_matrix) {
	for (int ii = 0; ii < num_nodes; ii++) {
		for (int jj = ii; jj < num_nodes; jj++) {
			if (ii == jj) {
				dist_matrix[ii * num_nodes + jj] = 0;
			} else {
				dist_matrix[ii * num_nodes + jj] = dist(nodes[ii], nodes[jj]);	// dist_matrix[ii][jj] = dist(ii, jj)
				dist_matrix[jj * num_nodes + ii] = dist_matrix[ii * num_nodes + jj];	//dist_matrix[jj][ii] = dist_matrix[ii][jj]
			}
		}
	}
	return dist_matrix;
}

//__device__
void print_matrix(int num_nodes, float* dist_matrix) {
	for (int ii = 0; ii < num_nodes; ii++) {
		for (int jj = ii; jj < num_nodes; jj++) {
			printf("%5f   ", dist_matrix[ii * num_nodes + jj]);
		}
		printf("\n");
	}
}

//__device__ 
void update_matrix_result(int num_iter, int num_nodes, int* closest, float* matrix, int* result) {
	result[num_iter * num_nodes] = num_iter;
}

//__device__
void find_closest(int num_nodes, int* closest, float* dist_matrix) {
	float closest_dist = 9999;
	for (int ii = 0; ii < num_nodes; ii++) {
		for (int jj = ii + 1; jj < num_nodes; jj++) {
			if (dist_matrix[ii * num_nodes + jj] == 0) {
				continue;
			}
			if (dist_matrix[ii * num_nodes + jj] < closest_dist) {
				closest[0] = ii;
				closest[1] = jj;
			}
		}
	}
}

//__global__ 
void hac(int num_nodes, node* nodes, int* result) {
	//__shared__
	float dist_matrix[num_nodes * num_nodes];
	//__shared__
	int closest[2];
	
	generate_dist_matrix(num_nodes, nodes, dist_matrix);	// compute dist matrix
	print_matrix(num_nodes, dist_matrix);
	
	int num_iter = 0;
	while (num_iter < num_nodes - 1) {			// while (# of cluster > 1) 
		find_closest(num_nodes, closest, dist_matrix);		// 	choose two clusters, a set of node(s), that have the closest distance
		update_matrix_result(num_iter, num_nodes, closest, dist_matrix, result);			// 	recompute dist matrix
									// 	every dist in one cluster = 0
									// 	dist outside the cluster = the shortest one between the node and an any node in the cluster
		num_iter++;
	}
}

/* 	TODO

//__host__ 
void call_hac(node* nodes, int num_nodes) {   				// memory alloc and hac execute
	node* d_in_nodes;						// for cudamalloc in 'nodes data'

	int* d_out_result;						// for cudamalloc out 'result'
	hipMalloc((node**) &d_in_nodes, num_nodes * sizeof(node));				
	hipMalloc((int**) &d_out_result, num_nodes * num_nodes * sizeof(int));
	hipMemcpy(d_in_nodes, nodes, num_nodes * sizeof(node), hipMemcpyHostToDevice);	// cuda memcopy h to d;
	
	hac<<<1, num_nodes>>>(num_nodes, nodes, d_out_result);				// hac <<< ?, ? >>> (N, in, out);
	
	hipMemcpy(result, d_out_result, num_nodes * num_nodes * sizeof(int), hipMemcpyDeviceToHost);	// cuda memcopy d to h 

	hipFree(d_in_nodes);		// cuda free (in)
	hipFree(d_out_result);		// cuda free (out)
}

*/

/*
void createDataset();
void printData(const Data& d);
__device__ double dist(const double* x, const double* y, int size);
*/

int main (int argc, char** argv) {
	if (argc != 2) {
		printf("Usage: %s <input file>", argv[0]);
		return 0;
	} else {
		node *nodes = NULL;
		int num_nodes = read_data(&nodes, argv[1]);
		if (num_nodes) {
			printf("Let's do CLUSTER!\n");
			//mem_alloc
			//__global__ void hac(&nodes);		// cuda
			
			free(nodes);
		} else {
			printf("empty file.\n");
		}
	}
	if (num_nodes) {
		printf("print the result!\n");
	}

	return 0;
}
