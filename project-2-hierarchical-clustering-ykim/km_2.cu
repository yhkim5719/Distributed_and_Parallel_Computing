#include <hip/hip_runtime.h>
#include <float.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#define N 64
#define K 3

#define THPERBLOCK 32
#define ITER 100

typedef struct Data {
	float* x;
        float* y;
} data;

data* read_data(const char* file) {
	data* d = NULL;
	FILE* f = fopen (file, "r");

/*
	int num_node;
	if (fscanf(f, "%5d\n", &num_node) == 0) {
		printf("read inputfile failed.\n");
		return NULL;
	}
	printf("num_node = %d\n", num_node);
*/	
	d = (data*) malloc(sizeof(data));
	d->x = (float*)malloc(N * sizeof(float));
	d->y = (float*)malloc(N * sizeof(float));
	for (int i = 0; i < N; i++) {
		fscanf(f, "%f %f\n", &(d->x[i]), &(d->y[i]));
	}

	printf("Point --------------------------------------\n");
	for (int i = 0; i < N; i++) {
		printf("%d: x = %f, y = %f\n", i, d->x[i], d->y[i]);
	}
	printf("\n");
	
	fclose(f);
	
	return d;
}		

data* read_cent(const char* file) {
	data* d = NULL;
	FILE* f = fopen (file, "r");

	d = (data*) malloc(sizeof(data));
	d->x = (float*)malloc(K * sizeof(float));
	d->y = (float*)malloc(K * sizeof(float));
	for (int i = 0; i < K; i++) {
		fscanf(f, "%f %f\n", &(d->x[i]), &(d->y[i]));
	}

	printf("Cluster location ---------------------------\n");
	for (int i = 0; i < K; i++) {
		printf("%d: x = %f, y = %f\n", i, d->x[i], d->y[i]);
	}
	printf("\n");
	
	fclose(f);
	
	return d;
}		

float dist_err(const float x, const float y, const float cx, const float cy) {
	return sqrt(pow(x - cx, 2.0) + pow(y - cy, 2.0));
}

float calculate_err(float* prev_cx, float* prev_cy, float* cx_result, float* cy_result) {
	float err = 0;
	for (int i = 0; i < K; i++) {
		err += dist_err(prev_cx[i], prev_cy[i], cx_result[i], cy_result[i]);
		prev_cx[i] = cx_result[i];
		prev_cy[i] = cy_result[i];
	}
	err /= K;

	return err;
}

__device__ float dist(const float x, const float y, const float cx, const float cy) {
	return sqrtf(powf(x - cx, 2.0) + powf(y - cy, 2.0));
}

__global__ void cluster_assign(const float* x, const float* y, float* cx, float* cy, int* cluster_label) {
	
	const unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (id >= N) {
		return;
	}

	float min_dist = 99999999;
	int closest_cent = 0;

	for (int i = 0; i < K; i++) {
		float distance = dist(x[id], y[id], cx[i], cy[i]);

		if (distance < min_dist) {
			min_dist = distance;
			closest_cent = i;
		}
	}

	cluster_label[id] = closest_cent;

}

__global__ void cent_update(const float* x, const float* y, float* cx, float* cy, int* cluster_label, int* cluster_size) {
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (id >= N) {
		return;
	}

	const int s_id = threadIdx.x;
	
	__shared__ float s_x[THPERBLOCK];
	__shared__ float s_y[THPERBLOCK];
	__shared__ int s_cluster_label[THPERBLOCK];

	s_x[s_id] = x[id];
	s_y[s_id] = y[id];
	s_cluster_label[s_id] = cluster_label[id];

	__syncthreads();

	if (s_id == 0) {
		float cent_x_sum[K] = { 0 };
		float cent_y_sum[K] = { 0 };
		int cent_cluster_size[K] = { 0 };

		for (int i = 0; i < blockDim.x; i++) {
			int cluster_id = s_cluster_label[i];
			cent_x_sum[cluster_id] += s_x[i];
			cent_y_sum[cluster_id] += s_y[i];
			cent_cluster_size[cluster_id]++;
		}

		for (int i = 0; i < K; i++) {
			atomicAdd(&cx[i], cent_x_sum[i]);
			atomicAdd(&cy[i], cent_y_sum[i]);
			atomicAdd(&cluster_size[i], cent_cluster_size[i]);
		}
	}
	
	__syncthreads();

	if (id < K) {
		cx[id] = cx[id] / cluster_size[id];
		cy[id] = cy[id] / cluster_size[id];
	}
}
	

void print_result (const float* cx, const float* cy, const int* label) {
	printf("\t");
	for (int i = 0; i < N; i++) {
		printf("%3d", i);
	}
	printf("\nLabel : ");
	for (int i = 0; i < N; i++) {
		printf("%3d", label[i]);
	}
	for (int i = 0; i < K; i++) {
		printf("\ncentroid %d\t :%5f %5f", i, cx[i], cy[i]);
	}
	printf("\n");
}

int main (int argc, char** argv) {
	if (argc != 3) {
		printf("Usage: %s <nodes file> <centroids file>\n", argv[0]);
		return 0;
	} else {
		data* d = read_data(argv[1]);
		data* c = read_cent(argv[2]);
		
		float* cx_result = (float*)malloc(K * sizeof(float));
		float* cy_result = (float*)malloc(K * sizeof(float));
		int* result = (int*) malloc(N * sizeof(int));
		
		printf("Let's do CLUSTER!\n");
			
		float* d_x;						// for cudamalloc in 'nodes data'
		float* d_y;
		int* d_cluster_label;
	
		float* d_cx;
		float* d_cy;
		float* d_cx_result;						// result centroid
		float* d_cy_result;
		int* d_cluster_size;

		hipMalloc((void**) &d_x, N * sizeof(float));				
		hipMalloc((void**) &d_y, N * sizeof(float));				
		hipMalloc((void**) &d_cluster_label, N * sizeof(int));				
	
		hipMalloc((void**) &d_cx, K * sizeof(float));				
		hipMalloc((void**) &d_cy, K * sizeof(float));				
		hipMalloc((void**) &d_cx_result, K * sizeof(float));				
		hipMalloc((void**) &d_cy_result, K * sizeof(float));				
		hipMalloc((void**) &d_cluster_size, K * sizeof(int));				
	
		hipMemcpy(d_x, d->x, N * sizeof(float), hipMemcpyHostToDevice);	// node memcpy
		hipMemcpy(d_y, d->y, N * sizeof(float), hipMemcpyHostToDevice);		// centroid memcpy
		hipMemcpy(d_cx, c->x, K * sizeof(float), hipMemcpyHostToDevice);	// node memcpy
		hipMemcpy(d_cy, c->y, K * sizeof(float), hipMemcpyHostToDevice);		// centroid memcpy
	
		int cur_iter = 1;
		
		float err = 1;
		float* prev_cx = (float*)malloc(K * sizeof(float));
		float* prev_cy = (float*)malloc(K * sizeof(float));
		for (int i = 0; i < K; i++) {
			prev_cx[i] = c->x[i];
			prev_cy[i] = c->y[i];
		}
			
		while (cur_iter < ITER) {
	
			cluster_assign<<<(N + THPERBLOCK - 1)/THPERBLOCK, THPERBLOCK>>>(d_x, d_y, d_cx, d_cy, d_cluster_label);
		
//			cudaMemcpy(cx_result, d_cx, K * sizeof(float), cudaMemcpyDeviceToHost);
//			cudaMemcpy(cy_result, d_cy, K * sizeof(float), cudaMemcpyDeviceToHost);

			
			hipMemset(d_cx, 0.0, K * sizeof(float));
			hipMemset(d_cy, 0.0, K * sizeof(float));
			hipMemset(d_cluster_size, 0, K * sizeof(int));
			
			cent_update<<<(N + THPERBLOCK - 1)/THPERBLOCK, THPERBLOCK>>>(d_x, d_y, d_cx, d_cy, d_cluster_label, d_cluster_size);
			
			hipMemcpy(cx_result, d_cx, K * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(cy_result, d_cy, K * sizeof(float), hipMemcpyDeviceToHost);
			
			for (int i = 0; i < K; i++) {
				printf("Iter %d:\tcent %d\t x:\t%f\ty:\t%f\n", cur_iter, i, cx_result[i], cy_result[i]);
			}
			printf("\n");

			err = calculate_err(prev_cx, prev_cy, cx_result, cy_result);
			if (err < 0.0001) {
				break;
			}
			cur_iter++;
		}
	
		hipMemcpy(result, d_cluster_label, N * sizeof(int), hipMemcpyDeviceToHost);	// cuda memcopy d to h 
//		cudaMemcpy(cx_result, d_cx_result, K * sizeof(float), cudaMemcpyDeviceToHost);
//		cudaMemcpy(cy_result, d_cy_result, K * sizeof(float), cudaMemcpyDeviceToHost);
	
		hipFree(d_x);
		hipFree(d_y);	
		hipFree(d_cluster_label);	
	
		hipFree(d_cx);
		hipFree(d_cy);
		hipFree(d_cx_result);	
		hipFree(d_cy_result);	
		hipFree(d_cluster_size);	
		
		printf("print the result!\n");
		
		print_result(cx_result, cy_result, result);
	
//		free(d);
//		free(c);
		free(cx_result);
		free(cy_result);
		free(result);
		free(prev_cx);
		free(prev_cy);
	}
	return 0;
}

/*
// TODO
void free_data(data* d) {
}
*/
