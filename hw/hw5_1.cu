#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void dkernel (unsigned* arr) {
	unsigned id = blockIdx.x * blockDim.x * blockDim.y * blockDim.z 
		    + threadIdx.z * blockDim.y * blockDim.x 
		    + threadIdx.y * blockDim.x 
		    + threadIdx.x;
	arr[id] = 0;
//	printf ("threadIdx. x, y, z = %d, %d, %d\n", threadIdx.x, threadIdx.y, threadIdx.z);
}

#define N 16
#define M 1
#define L 1

int main() {
	dim3 block(N, M, L);
	unsigned *arr, *harr;
	printf ("Size of Array = %d\n", N * M * L);
	hipMalloc (&arr, N * M * sizeof(unsigned));
	harr = (unsigned *)malloc (N * M * L * sizeof(unsigned));
	dkernel<<<1, block>>> (arr);
	hipMemcpy (harr, arr, N * M * L * sizeof(unsigned), hipMemcpyDeviceToHost);
	for (unsigned ii = 0; ii < N; ii++) {
		for (unsigned jj = 0; jj < M; jj++) {
			for (unsigned kk = 0; kk < L; kk++) {
				printf ("%d ", harr[ii * M * L + jj * L + kk]);
			}
			printf("\n");
		}
		printf("\n");
	}
	return 0;
}
