#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 32

void printMatrix (unsigned* matrix) {
	for (unsigned i = 0; i < N * N; i++) {
		printf(" %u ", matrix[i]);
		if (i % N == (N-1)) {
			printf("\n");
		}
	}
} 

void createMatrix(unsigned* matrix) {
	for (unsigned i = 0; i < N; i++) {
		for (unsigned j = 0; j < N; j++) {
			if (i == j) {
				matrix[i * N + j] = i + 1;
			} else {
				matrix[i * N + j] = 0;
			}
		}
	}	
}

__global__ void square (unsigned* matrix, unsigned* result, unsigned matrixSize) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	for (unsigned jj = 0; jj < matrixSize; jj++) {
		for (unsigned kk = 0; kk < matrixSize; kk++) {
			result[id * matrixSize + jj] += matrix[id * matrixSize + kk] * matrix[kk * matrixSize + jj];
		}
	}
}

__host__ void call_sqr (unsigned* h_in_matrix, unsigned* h_out_matrix) {
//	unsigned n = N;
	unsigned *d_in_matrix, *d_out_matrix;
	hipMalloc((void **) &d_in_matrix, N * N * sizeof(unsigned));
	hipMalloc((void **) &d_out_matrix, N * N * sizeof(unsigned));
	hipMemcpy(d_in_matrix, h_in_matrix, N * N * sizeof(unsigned), hipMemcpyHostToDevice);
	
	square<<<1, N>>>(d_in_matrix, d_out_matrix, N);
	
	hipMemcpy(h_out_matrix, d_out_matrix, N * N * sizeof(unsigned), hipMemcpyDeviceToHost);

	hipFree(d_in_matrix);
	hipFree(d_out_matrix);	
}

int main() {
	unsigned* matrix;
	unsigned* result;
	matrix = (unsigned*)malloc(N * N * sizeof(unsigned));
	result = (unsigned*)malloc(N * N * sizeof(unsigned));
	createMatrix(matrix);
	call_sqr (matrix, result);
	printMatrix(result);
	free(matrix);
	free(result);

	return 0;
}
