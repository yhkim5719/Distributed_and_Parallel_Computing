#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void dkernel (unsigned* arr) {
	unsigned id = blockIdx.x * blockDim.x * blockDim.y * blockDim.z 
		    + threadIdx.z * blockDim.y * blockDim.x 
		    + threadIdx.y * blockDim.x 
		    + threadIdx.x;
	arr[id] = 0;
//	printf ("threadIdx. x, y, z = %d, %d, %d\n", threadIdx.x, threadIdx.y, threadIdx.z);
}

__global__ void add (unsigned* arr) {
	unsigned id = blockIdx.x * blockDim.x * blockDim.y * blockDim.z 
		    + threadIdx.z * blockDim.y * blockDim.x 
		    + threadIdx.y * blockDim.x 
		    + threadIdx.x;
	arr[id] += id;
}


#define N 16	// blockDim.x
#define M 8	// blockDim.y
#define L 8	// blockDim.z

int main() {
	dim3 block(N, M, L);
	unsigned *arr, *harr;
	printf ("Size of Array = %d\n", N * M * L);
	hipMalloc (&arr, N * M * L * sizeof(unsigned));
	harr = (unsigned *)malloc (N * M * L * sizeof(unsigned));
	dkernel<<<1, block>>> (arr);
	add<<<1, block>>> (arr);
	hipMemcpy (harr, arr, N * M * L * sizeof(unsigned), hipMemcpyDeviceToHost);
	for (unsigned ii = 0; ii < N; ii++) {
		for (unsigned jj = 0; jj < M; jj++) {
			for (unsigned kk = 0; kk < L; kk++) {
				printf ("%d ", harr[ii * M * L + jj * L + kk]);
			}
			printf("\n");
		}
		printf("\n");
	}
	return 0;
}
